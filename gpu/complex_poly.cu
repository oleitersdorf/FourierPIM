#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <chrono>

#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipfft/hipfftXt.h>

// FP16 if specified, otherwise FP32
#ifdef HALF_PRECISION
typedef half2 Complex;
const hipDataType DTYPE = HIP_C_16F;
#else
typedef float2 Complex;
const hipDataType DTYPE = HIP_C_32F;
#endif

static __device__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMul(Complex *, const Complex *, int);

/**
 * Performs the experiment with the given parameters
 * @param vectorSize the vector size (n) used for the experiment
 * @param batchSize the batch size (m) used for the experiment
 * @param numIterations the number of iterations in the experiment (used for average iteration time)
 */
void runTest(long long vectorSize, long long batchSize, long long numIterations){

    // Events used for timing
    float time;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Allocate host memory
    Complex* hx_data = new Complex[vectorSize * batchSize];
    for (long long i = 0; i < vectorSize * batchSize; i++)
        hx_data[i].x = rand() / (float)RAND_MAX, hx_data[i].y = rand() / (float)RAND_MAX;
    Complex* hy_data = new Complex[vectorSize * batchSize];
    for (long long i = 0; i < vectorSize * batchSize; i++)
        hy_data[i].x = rand() / (float)RAND_MAX, hy_data[i].y = rand() / (float)RAND_MAX;

    // Allocate device memory
    Complex* dx_data;
    checkCudaErrors(hipMalloc(&dx_data, vectorSize * batchSize * sizeof(Complex)));
    Complex* dy_data;
    checkCudaErrors(hipMalloc(&dy_data, vectorSize * batchSize * sizeof(Complex)));

    // Create FFT plan
    hipfftHandle plan;
    checkCudaErrors(hipfftCreate(&plan));
    size_t ws = 0;
    checkCudaErrors(hipfftXtMakePlanMany(plan, 1, &vectorSize, NULL, 1, 1, DTYPE, NULL, 1, 1, DTYPE, batchSize, &ws, DTYPE));

    // Copy to device
    checkCudaErrors(hipMemcpy(dx_data, hx_data, vectorSize * batchSize * sizeof(Complex), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dy_data, hy_data, vectorSize * batchSize * sizeof(Complex), hipMemcpyHostToDevice));

    // Execute polynomial multiplication
    checkCudaErrors(hipEventRecord(start, 0));

    std::cout << "START" << std::endl;

    for (int i = 0; i < numIterations; i++) {

        // Perform forward FFTs
        checkCudaErrors(hipfftXtExec(plan, (void *) dx_data, (void *) dx_data, HIPFFT_FORWARD));
        checkCudaErrors(hipfftXtExec(plan, (void *) dy_data, (void *) dy_data, HIPFFT_FORWARD));

        // Perform element-wise multiplication
        ComplexPointwiseMul<<<(vectorSize * batchSize) / 256, 256>>>(dx_data, dy_data, vectorSize * batchSize);
        getLastCudaError("Kernel execution failed [ ComplexPointwiseMul ]");

        // Perform inverse FFT
        checkCudaErrors(hipfftXtExec(plan, (void *) dx_data, (void *) dx_data, HIPFFT_BACKWARD));

    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    std::cout << "STOP" << std::endl;

    // Print results
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));
    std::cout << time / numIterations << std::endl;

    // Cleanup
    checkCudaErrors(hipfftDestroy(plan));
    checkCudaErrors(hipFree(dx_data));
    checkCudaErrors(hipFree(dy_data));
    checkCudaErrors(hipDeviceReset());

}

static __device__ inline Complex ComplexMul(Complex a, Complex b) {
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

/**
 * Performs a pointwise multiplication
 */
static __global__ void ComplexPointwiseMul(Complex *a, const Complex *b, int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = ComplexMul(a[i], b[i]);
}

/**
 * Parses the given parameters and executes the experiment.
 * Execution format: ./fft VECTOR_SIZE BATCH_SIZE NUM_ITERATIONS
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv){

    assert(argc == 4);

    // Parse VECTOR_SIZE
    long long vectorSize = atol(argv[1]);

    // Parse BATCH_SIZE
    long long batchSize = atol(argv[2]);

    // Parse NUM_ITERATIONS
    long long numIterations = atol(argv[3]);

    // Run the experiment
    runTest(vectorSize, batchSize, numIterations);

    return 0;

}
