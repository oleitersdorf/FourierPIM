#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <chrono>

#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipfft/hipfftXt.h>

// FP16 if specified, otherwise FP32
#ifdef HALF_PRECISION
typedef half Real;
typedef half2 Complex;
const hipDataType CDTYPE = HIP_C_16F;
const hipDataType RDTYPE = HIP_R_16F;
#else
typedef float Real;
typedef float2 Complex;
const hipDataType CDTYPE = HIP_C_32F;
const hipDataType RDTYPE = HIP_R_32F;
#endif

static __device__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMul(Complex *, const Complex *, int);

/**
 * Performs the experiment with the given parameters
 * @param vectorSize the vector size (n) used for the experiment
 * @param batchSize the batch size (m) used for the experiment
 * @param numIterations the number of iterations in the experiment (used for average iteration time)
 */
void runTest(long long vectorSize, long long batchSize, long long numIterations){

    // Events used for timing
    float time;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Allocate host memory
    Real* hx_data = new Real[vectorSize * batchSize];
    for (long long i = 0; i < vectorSize * batchSize; i++)
        hx_data[i] = rand() / (float)RAND_MAX;
    Real* hy_data = new Real[vectorSize * batchSize];
    for (long long i = 0; i < vectorSize * batchSize; i++)
        hy_data[i] = rand() / (float)RAND_MAX;

    // Allocate device memory
    Complex* dx_data;
    checkCudaErrors(hipMalloc(&dx_data, vectorSize * batchSize * sizeof(Complex)));
    Complex* dy_data;
    checkCudaErrors(hipMalloc(&dy_data, vectorSize * batchSize * sizeof(Complex)));

    // Create FFT plan
    hipfftHandle R2CPlan;
    checkCudaErrors(hipfftCreate(&R2CPlan));
    size_t R2Cws = 0;
    checkCudaErrors(hipfftXtMakePlanMany(R2CPlan, 1, &vectorSize, NULL, 1, 1, RDTYPE, NULL, 1, 1, CDTYPE, batchSize, &R2Cws, CDTYPE));
    hipfftHandle C2CPlan;
    size_t C2Cws = 0;
    checkCudaErrors(hipfftCreate(&C2CPlan));
    checkCudaErrors(hipfftXtMakePlanMany(C2CPlan, 1, &vectorSize, NULL, 1, 1, CDTYPE, NULL, 1, 1, CDTYPE, batchSize, &C2Cws, CDTYPE));

    // Copy to device
    checkCudaErrors(hipMemcpy(dx_data, hx_data, vectorSize * batchSize * sizeof(Real), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dy_data, hy_data, vectorSize * batchSize * sizeof(Real), hipMemcpyHostToDevice));

    // Execute polynomial multiplication
    checkCudaErrors(hipEventRecord(start, 0));

    std::cout << "START" << std::endl;

    for (int i = 0; i < numIterations; i++) {

        // Perform forward FFTs
        checkCudaErrors(hipfftXtExec(R2CPlan, (void *) dx_data, (void *) dx_data, HIPFFT_FORWARD));
        checkCudaErrors(hipfftXtExec(R2CPlan, (void *) dy_data, (void *) dy_data, HIPFFT_FORWARD));

        // Perform element-wise multiplication
        ComplexPointwiseMul<<<(vectorSize * batchSize) / 256, 256>>>(dx_data, dy_data, vectorSize * batchSize);
        getLastCudaError("Kernel execution failed [ ComplexPointwiseMul ]");

        // Perform inverse FFT
        checkCudaErrors(hipfftXtExec(C2CPlan, (void *) dx_data, (void *) dx_data, HIPFFT_BACKWARD));

    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));

    std::cout << "STOP" << std::endl;

    // Print results
    checkCudaErrors(hipEventElapsedTime(&time, start, stop));
    std::cout << time / numIterations << std::endl;

    // Cleanup
    checkCudaErrors(hipfftDestroy(R2CPlan));
    checkCudaErrors(hipfftDestroy(C2CPlan));
    checkCudaErrors(hipFree(dx_data));
    checkCudaErrors(hipFree(dy_data));
    checkCudaErrors(hipDeviceReset());

}

static __device__ inline Complex ComplexMul(Complex a, Complex b) {
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

/**
 * Performs a pointwise multiplication
 */
static __global__ void ComplexPointwiseMul(Complex *a, const Complex *b, int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    a[i] = ComplexMul(a[i], b[i]);
}

/**
 * Parses the given parameters and executes the experiment.
 * Execution format: ./fft VECTOR_SIZE BATCH_SIZE NUM_ITERATIONS
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv){

    assert(argc == 4);

    // Parse VECTOR_SIZE
    long long vectorSize = atol(argv[1]);

    // Parse BATCH_SIZE
    long long batchSize = atol(argv[2]);

    // Parse NUM_ITERATIONS
    long long numIterations = atol(argv[3]);

    // Run the experiment
    runTest(vectorSize, batchSize, numIterations);

    return 0;

}
